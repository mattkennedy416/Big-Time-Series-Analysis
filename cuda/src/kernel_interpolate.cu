
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

void __global__ kernel_linearInterpolation(float* array_device, int* rowArray, int rowArrayLength, int* colArray, int colArrayLength, int totalCols, int totalRows, float* originalTimes, int originalTimesLength, float* newTimes, int newTimesLength, float* results_device)
{
	
	// where original times and new times should probably be unix time? but technically won't matter probably
	
	int n = blockIdx.x * blockDim.x + threadIdx.x;
	int m = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (n < newTimesLength && m < colArrayLength)
	{
		// so we created a row-wise thread per element in the newTimes, with m being number of columns
		
		float newTime = newTimes[n];
		int resultsInd = n*colArrayLength + m;
		
		
		// need to potentially handle the first and last points in a special way ,,,
		if (n==0 && newTimes[0] < originalTimes[0])
		{
			// lets say if newTimes[0] is prior to originalTimes[0] less than the distance from originalTimes[1] to originalTimes[2], we'll accept it
			// otherwise call it a NaN
			if ( originalTimes[0] - newTimes[0] < originalTimes[1] - originalTimes[0] )
			{
				// continue slope from points 0 to 1?
				// ie assume there's a previous point at same slope and same sample rate
				int arrayInd1 = (0)*totalCols + m;
				int arrayInd2 = (1)*totalCols + m;
				float valueChange = array_device[arrayInd2] - array_device[arrayInd1];
				float timeDiff = originalTimes[1] - originalTimes[0];
				
				float pseudoPointTime = originalTimes[0] - timeDiff;
				float pseudoPointValue = array_device[arrayInd1] - valueChange;
				
				float perc = (newTime - pseudoPointTime) / timeDiff;
				
				results_device[resultsInd] = pseudoPointValue + perc*valueChange;
				
			}
			else
				results_device[resultsInd] = NAN;
			
			return;			
		}
		
		// ================================================
		// ================================================
		// ===== DO THIS SAME LOGIC FOR LAST POINT? =======
		// ================================================
		// ================================================
		
		// we now need to find the points before and after ...
		// do we need to iterate through the entire originalTimes vector or is there some way we can approximate the starting location?
		
		for (int originalTimeLoc=0; originalTimeLoc<originalTimesLength-1; originalTimeLoc++)
		{
//			if (newTime > originalTimes[originalTimeLoc+1]) // gone too far and haven't found anything
//			{
//				results_device[resultsInd] = NAN;
//				return;
//			}
			
			
			if ( originalTimes[originalTimeLoc] <= newTime && newTime <= originalTimes[originalTimeLoc+1] )
			{
				printf("Found our location: %f < %f < %f\n", originalTimes[originalTimeLoc], newTime, originalTimes[originalTimeLoc+1]);
				// alright this is our point
				int arrayInd1 = originalTimeLoc*totalCols + m;
				int arrayInd2 = (originalTimeLoc+1)*totalCols + m;				
				
				float valueChange = (array_device[arrayInd2] - array_device[arrayInd1]);
				
				// we need to figure out what percentage the newTime is between the two points
				float origTime1 = originalTimes[originalTimeLoc];
				float origTime2 = originalTimes[originalTimeLoc+1];
				
				float perc = (newTime - origTime1) / (origTime2 - origTime1);
				
				results_device[resultsInd] = array_device[arrayInd1] + perc*valueChange;
				
				return;			
				
			}
		}
		
	
	}
	
}


